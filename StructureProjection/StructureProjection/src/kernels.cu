#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include<stdio.h>
#include<kernels.h>

//using namespace std;
//************************************************************************************************************************
//************************************************************************************************************************
__global__ void ProjectionGPU(float *d_Label, int *d_im_EPID, float *d_sourcePoint,  float *d_resolution, float d_BeamAngle, int NbRows, int NbCols, int NbSlices, int d_PanelNbPixels, float d_PixDimEPID, float d_TableAngle, int Xmin,int Xmax,int Ymin,int Ymax,int Zmin,int Zmax,float offsetX_dicom,float offsetY_dicom,float offsetZ_dicom)
{
	
    int x_target = blockIdx.x;
    int y_target = threadIdx.x;

	if (x_target<d_PanelNbPixels && y_target<d_PanelNbPixels){

		
    float ai,aj;
    float targetpoint0, targetpoint1,targetpoint2;
    float axy,bxy,axz,bxz,ayx,ayz,byx,byz,azx,azy,bzx,bzy;
    float x_mm,y_mm,z_mm;
    int y_intersectX,z_intersectX,x_intersectY,z_intersectY,x_intersectZ,y_intersectZ;
    int xL,yL,zL;
        
    ai = (x_target-d_PanelNbPixels/2+0.5)*d_PixDimEPID;
    aj = (y_target-d_PanelNbPixels/2+0.5)*d_PixDimEPID;
        
    float PI = 3.14159;
        
    targetpoint0 = cos(d_BeamAngle*PI/180)*ai-sin(d_BeamAngle*PI/180)*600;
    targetpoint1 = sin(d_BeamAngle*PI/180)*ai+cos(d_BeamAngle*PI/180)*600;
    targetpoint2 = aj;
        
    axy = (targetpoint1-d_sourcePoint[1])/(targetpoint0-d_sourcePoint[0]);
    bxy = d_sourcePoint[1]-axy*d_sourcePoint[0];
    axz = (targetpoint2-d_sourcePoint[2])/(targetpoint0-d_sourcePoint[0]);
    bxz = d_sourcePoint[2]-axz*d_sourcePoint[0];

    for(int x=Xmin-1;x<(Xmax+2);x++){
        x_mm = x*d_resolution[0] - offsetX_dicom;
        y_intersectX = (int)round(((x_mm*axy+bxy)+offsetY_dicom)/d_resolution[1]);
        z_intersectX = (int)round(((x_mm*axz+bxz)+offsetZ_dicom)/d_resolution[2]);
        xL=x;

                
        if ((xL >= 0) and (xL < NbRows) and (y_intersectX >= 0) and (y_intersectX < NbCols) and (z_intersectX >= 0) and (z_intersectX < NbSlices)){
                    
            if(d_Label[z_intersectX*NbRows*NbCols+y_intersectX*NbCols+xL]>0){
                d_im_EPID[x_target*d_PanelNbPixels+y_target] = 1;
                x = Xmax+1;
            }
        }
    }
    
    if(d_im_EPID[x_target*d_PanelNbPixels+y_target] ==0){
        ayx = (targetpoint0-d_sourcePoint[0])/(targetpoint1-d_sourcePoint[1]);
        byx = d_sourcePoint[0]-ayx*d_sourcePoint[1];
        ayz = (targetpoint2-d_sourcePoint[2])/(targetpoint1-d_sourcePoint[1]);
        byz = d_sourcePoint[2]-ayz*d_sourcePoint[1];
                
                
        for(int y=Ymin-1;y<(Ymax+2);y++){
            y_mm = y*d_resolution[1] - offsetY_dicom;
            x_intersectY = (int)round(((y_mm*ayx+byx)+offsetX_dicom)/d_resolution[0]);
            z_intersectY = (int)round(((y_mm*ayz+byz)+offsetZ_dicom)/d_resolution[2]);
            yL=y;
                    
            if ((yL >= 0) and (yL < NbCols) and (x_intersectY >= 0) and (x_intersectY < NbRows) and (z_intersectY >= 0) and (z_intersectY < NbSlices)){
                        
                if(d_Label[z_intersectY*NbRows*NbCols+yL*NbCols+x_intersectY]>0){
                    d_im_EPID[x_target*d_PanelNbPixels+y_target] = 1;
                    y = Ymax+1;
                }
            }
        }
    }
    
    if(d_im_EPID[x_target*d_PanelNbPixels+y_target] ==0){
        azx = (targetpoint0-d_sourcePoint[0])/(targetpoint2-d_sourcePoint[2]);
        bzx = d_sourcePoint[0]-azx*d_sourcePoint[2];
        azy = (targetpoint1-d_sourcePoint[1])/(targetpoint2-d_sourcePoint[2]);
        bzy = d_sourcePoint[1]-azy*d_sourcePoint[2];
               
        for(int z=Zmin-1;z<(Zmax+2);z++){
            z_mm = z*d_resolution[2] - offsetZ_dicom;
            x_intersectZ = (int)round(((z_mm*azx+bzx)+offsetX_dicom)/d_resolution[0]);
            y_intersectZ = (int)round(((z_mm*azy+bzy)+offsetY_dicom)/d_resolution[1]);
            zL=z;
                    
            if ((y_intersectZ >= 0) and (y_intersectZ < NbCols) and (x_intersectZ >= 0) and (x_intersectZ < NbRows) and (zL >= 0) and (zL < NbSlices)){
                        
                if(d_Label[zL*NbRows*NbCols+y_intersectZ*NbCols+x_intersectZ]>0){
                    d_im_EPID[x_target*d_PanelNbPixels+y_target] = 1;
                    z = Zmax+1;
                }
            }
        }
    }
        
        
    }

}

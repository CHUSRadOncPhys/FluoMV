#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <cstdlib>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "kernels.h"


using namespace std;

//************************************************************************************************************************************************
// This code projects the ROIs selected on a virtual EPID for every 0.5 degree and creates the Centroid.txt and the Results.txt files. Centroid.txt contains
//  the mean position of each projection and Results.txt contains the position of each points forming the contours of the projections.
//************************************************************************************************************************************************
int main (int argc, char **argv) 
{
    //~ time_t timer;
    //~ cout <<time(&timer)<<endl;
    //~ sleep(2);
    //~ cout <<time(&timer)<<endl;
    
    float *sourcePoint = new float[3];
    float *resolution = new float[3];
    float *isocenter = new float[3];
    float *refPointCT = new float[3];
    int *sens = new int[3];
    
    string PATH = argv[1];
    isocenter[0] = atof(argv[2]);
    isocenter[1] = atof(argv[3]);
    isocenter[2] = atof(argv[4]);
    sens[0] = atoi(argv[5]);
    sens[1] = atoi(argv[6]);
    sens[2] = atoi(argv[7]);
    string Patient_id = argv[8];
    resolution[0] = atof(argv[9]);
    resolution[1] = atof(argv[10]);
    resolution[2] = atof(argv[11]);
    refPointCT[0] = atof(argv[12]);
    refPointCT[1] = atof(argv[13]);
    refPointCT[2] = atof(argv[14]);
    int NbOfROIs = atoi(argv[15]);
    int NbRows = atoi(argv[16]);
    int NbCols = atoi(argv[17]);
    int NbSlices = atoi(argv[18]);
    string ROIName;
    struct timeval tp;
    long int start;
    long int end;
    int NbAngles = 720;
    
    int Xmin; int Xmax; int Ymin; int Ymax; int Zmin; int Zmax;
    
    int PanelNbPixels = 512;
    float PixDimEPID = 0.252*1024/PanelNbPixels*1.6;

    float BeamAngle = 0.0;
    float TableAngle = 0.0;
    float PI = 3.14159;
    
    int *Label = new int [NbSlices*NbRows*NbCols];
    int *im_EPID = new int [PanelNbPixels*PanelNbPixels];
    int *Panel = new int[PanelNbPixels*PanelNbPixels*NbAngles];
    
    for (int i = 0; i < PanelNbPixels*PanelNbPixels*NbAngles; i++) {
            Panel[i] = 0;
        }
    
    ifstream RoiFile;
    RoiFile.open("./listROI.txt");
    
    ofstream Centroid("./StructureProjection/Centroid.txt");
    ofstream Results("./StructureProjection/Results.txt");
    Results << NbOfROIs << "\t" << endl;
    Results << NbAngles << endl;
        
        
    for(int N=0; N<NbOfROIs; N++){
        
        RoiFile>>ROIName;
        RoiFile>>Xmin;RoiFile>>Xmax;RoiFile>>Ymin;RoiFile>>Ymax;RoiFile>>Zmin;RoiFile>>Zmax;
        
        Results << ROIName << endl;
        Centroid << ROIName << endl;
        
        streampos size =NbSlices*NbRows*NbCols;
        char * memblock; memblock = new char [size];
        
        
        
        ifstream myfile(("./ROIs/"+Patient_id+"."+ROIName).c_str(),ios::in|ios::binary);
        if (myfile.is_open()){
            myfile.read (memblock, size);
            myfile.close();
        }
        else{cout << "Unable to open file"<<endl;}
        int val;
        int pos = 0;
        for (int i = 0; i < NbSlices; i++) {
            for (int j = 0; j < NbRows; j++) {
                for (int k = 0; k < NbCols; k++) {
                    
                    
                    val = *(unsigned char *)&memblock[pos];
                    
                    Label[i*NbRows*NbCols+j*NbCols+k] = (int)val;
                    pos = pos + 1;
                }
            }
        }
        
        myfile.close();    
        delete[] memblock;
        

        
        //~ start = std::chrono::system_clock::now();
        
        hipError_t err;
        size = NbSlices*NbRows*NbCols * sizeof(int);
        float *d_Label = NULL;
        err = hipMalloc((void **)&d_Label, size);
        err = hipMemcpy(d_Label, Label, size, hipMemcpyHostToDevice);
        
        size = 3 * sizeof(float);
        float *d_resolution = NULL;
        err = hipMalloc((void **)&d_resolution, size);
        err = hipMemcpy(d_resolution, resolution, size, hipMemcpyHostToDevice);

        
        float offsetX_dicom = (isocenter[0] - refPointCT[0])*sens[0];
        float offsetY_dicom = (isocenter[1] - refPointCT[1])*sens[1];
        float offsetZ_dicom = (isocenter[2] - refPointCT[2])*sens[2]+(NbSlices-1)*resolution[2];
        
        gettimeofday(&tp, NULL);
        start = tp.tv_sec * 1000 + tp.tv_usec / 1000;
        
        int threadsPerBlock = PanelNbPixels;
        int blocksPerGrid = PanelNbPixels;
        
        
        
        for(int z=0;z<NbAngles;z++){
        
            BeamAngle = (float)(z)*0.5;
            
            
            sourcePoint[0] = 1000 * sin(BeamAngle*PI / 180.0);
            sourcePoint[1] = -1000 * cos(BeamAngle*PI / 180.0);
            sourcePoint[2] = 0.0;
            
            for (int i = 0; i < PanelNbPixels*PanelNbPixels; i++) {
                im_EPID[i] = 0;
            }
            
            size = 3 * sizeof(float);
            float *d_sourcePoint = NULL;
            err = hipMalloc((void **)&d_sourcePoint, size);
            err = hipMemcpy(d_sourcePoint, sourcePoint, size, hipMemcpyHostToDevice);
            
            size = PanelNbPixels*PanelNbPixels*sizeof(int);
            int *d_im_EPID = NULL;
            err = hipMalloc((void **)&d_im_EPID, size);
            err = hipMemcpy(d_im_EPID, im_EPID, size, hipMemcpyHostToDevice);

            ProjectionGPU<< < blocksPerGrid, threadsPerBlock >> >(d_Label, d_im_EPID, d_sourcePoint, d_resolution, BeamAngle, NbRows, NbCols, NbSlices, PanelNbPixels, PixDimEPID, TableAngle, Xmin, Xmax, Ymin, Ymax, Zmin, Zmax, offsetX_dicom, offsetY_dicom, offsetZ_dicom);
            
            size = PanelNbPixels*PanelNbPixels* sizeof(int);
            err = hipMemcpy(im_EPID, d_im_EPID, size, hipMemcpyDeviceToHost);
            //~ cout <<"err: "<< err << endl;
            float NbOfPoints=0;
            float mean_i = 0;
            float mean_j = 0;
            for(int i=0;i<PanelNbPixels;i++){
                for(int j=0;j<PanelNbPixels;j++){
                    if(im_EPID[i*PanelNbPixels + j] ==1){
                        NbOfPoints = NbOfPoints + 1;
                        mean_i = mean_i +i;
                        mean_j = mean_j +j;
                        if(i==0 or j==0 or i==PanelNbPixels-1 or j==PanelNbPixels-1){
                            Panel[z*PanelNbPixels*PanelNbPixels+(PanelNbPixels-1-j)*PanelNbPixels+i] = 1;
                        }
                        else{
                            if(im_EPID[(i+1)*PanelNbPixels + j]==1 and im_EPID[(i-1)*PanelNbPixels + j]==1 and im_EPID[i*PanelNbPixels + j+1]==1 and im_EPID[i*PanelNbPixels + j-1]==1 and im_EPID[(i+1)*PanelNbPixels + j+1]==1 and im_EPID[(i-1)*PanelNbPixels + j+1]==1 and im_EPID[(i+1)*PanelNbPixels + j-1]==1 and im_EPID[(i-1)*PanelNbPixels + j-1]==1){
                                Panel[z*PanelNbPixels*PanelNbPixels+(PanelNbPixels-1-j)*PanelNbPixels+i] = 0;  
                            }
                            else{Panel[z*PanelNbPixels*PanelNbPixels+(PanelNbPixels-1-j)*PanelNbPixels+i] = 1;}
                        }
                    }
                    else{Panel[z*PanelNbPixels*PanelNbPixels+(PanelNbPixels-1-j)*PanelNbPixels+i] = 0;}
                }
            }
            
            hipFree(d_sourcePoint);
            hipFree(d_im_EPID);
            
            mean_i = mean_i/NbOfPoints;
            mean_j = mean_j/NbOfPoints;
            Centroid << BeamAngle << "\t" << mean_i << "\t" << mean_j <<endl;
        }
        
        for (int z = 0; z < NbAngles; z++){
            vector<int> Contour_X;
            vector<int> Contour_Y;
                
            for (int x = 0; x < PanelNbPixels; x++){
                for (int y = 0; y < PanelNbPixels; y++){
                    if (Panel[z*PanelNbPixels*PanelNbPixels + y * PanelNbPixels + x] == 1){
                        Contour_X.push_back(x);
                        Contour_Y.push_back(y);
                    }
                }
            }
            
            for(int v=0;v<Contour_X.size();v++){
                    Results << Contour_X.at(v)*256.0/PanelNbPixels <<"\t" << Contour_Y.at(v)*256.0/PanelNbPixels <<"\t";
            }
            
            Results << endl;
            Contour_X.clear();
            Contour_Y.clear();
            
        }

        hipFree(d_Label);
        hipFree(d_resolution);    
        
        cout<<ROIName<<endl;
        
    }
    Centroid.close();
    Results.close();
    RoiFile.close();
        
    delete[] sourcePoint; sourcePoint = NULL;
    delete[] resolution; resolution = NULL;
    delete[] isocenter; isocenter = NULL;
    delete[] refPointCT; refPointCT = NULL;
    delete[] sens; sens = NULL;
    delete[] Label; Label = NULL;
    delete[] im_EPID; im_EPID = NULL;
    delete[] Panel; Panel = NULL;
    

    gettimeofday(&tp, NULL);
    end = tp.tv_sec * 1000 + tp.tv_usec / 1000;
    cout<<end-start<<endl;
    
    
    
    cout<<"Fin du programme"<<endl;
    return 0;
}
//=============================================================================================================================================